#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <string.h>

#ifndef N
#include "w15_10.h"
#endif



#include "cudaemu.h"
#include "cudacommon.h"

#define REPEAT 1

#define CONST


typedef unsigned long long flag_t;
typedef unsigned char edge_t;
typedef unsigned char node_t;
typedef unsigned dist_t;
typedef float flow_t;


#ifdef CONST
__constant__ node_t e[2*E];           //Edge
__constant__ flow_t C[N*N];           //Capacity
__constant__ flow_t D[N*N];           //Demand
__constant__ dist_t w[N*N];           //Weight
#endif

node_t* e_host;             //Edge
flow_t* C_host;             //Capacity
flow_t* D_host;             //Demand
dist_t* w_host;             //Weight


#ifndef __NDEBUG__



inline void printBitStream(flag_t b){
    bool first = 1;
    for(edge_t k = 0; k < E; k++){
        if( ( b >> k ) & 1 ){ if(first){ printf("%d", k); first = 0; }else{ printf(", %d", k); } }
    }
}

inline void printTable(flag_t* t){
    for (node_t i = 0; i < N; i++) {
        for (node_t j = 0; j < N; j++) {
            printf(" ["); printBitStream( t[i*N+j] ); printf("] ");
        }
        printf("\n");
    }
}


#endif




__global__ void gpu_ecmpWarshallfloydEdge(edge_t k, node_t x, node_t y, dist_t* d, flag_t* p){
    
    node_t i = threadIdx.x + blockIdx.x * blockDim.x;
    node_t j = threadIdx.y + blockIdx.y * blockDim.y;
    
    if(i<N && j<N){

        dist_t sub = d[i*N+x] + w[k] + d[y*N+j];
        if( sub < d[i*N+j] ) {
            d[i*N+j] = sub;
            p[i*N+j] = ((flag_t)1 << k);
            //printf("upd %d - %d via %d [ ", i, j, k); printBitStream(p[i*N+j]); printf("]\n");
        }else if( sub == d[i*N+j] ) {
            p[i*N+j] |= ((flag_t)1 << k);
            //printf("add %d - %d via %d [ ", i, j, k); printBitStream(p[i*N+j]); printf("]\n");
        }
    }
}






////__device__
flag_t getNeighborEdges(flag_t sub, const node_t s, const flag_t* p, const node_t* e){
    flag_t tmp = 0;
    for(edge_t k = 0; k < E; k++ ){ if( ( sub >> k ) & 1 ){       //only works where flag_t up
        const node_t near = e[k*2];
        //printf("near = %d\n", near);
        if(near == s){
            //printf("this is source node\n");
            tmp |= ((flag_t)1 << k);
        }else{
            //printf("follow again from %d to %d\n", s, near);
            tmp |= getNeighborEdges(p[s*N+near], s, p, e);
        }
    } }
    return tmp;
}


////__device__
void traceAndAlloc(const node_t h, const node_t d, const flag_t* n, int rate, const flow_t traffic, flow_t* f, node_t* e){
    
    if (h != d) {
        
        char branch = 0;
        for(edge_t k = 0; k < E; k++ ){
            if( ( n[h*N+d] >> k ) & 1 ){ branch++; }
        }
        rate *= branch;
        
        for(edge_t k = 0; k < E; k++ ){
            if( ( n[h*N+d] >> k ) & 1 ){
                node_t next = e[k*2+1];
                f[k] += traffic / rate;
                traceAndAlloc(next, d, n, rate, traffic, f, e);
            }
        }
        
    }
    
}




int main(){
    
    
    printf("sizeof = node_t:%lu, flow_t:%lu, dist_t:%lu, flag_t:%lu\n", sizeof(node_t), sizeof(flow_t), sizeof(dist_t), sizeof(flag_t));

    //property
    hipDeviceProp_t  prop;    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    int mp = prop.multiProcessorCount;
    
    
    //time
    hipEvent_t start, wf;     float ms=0.0f;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &wf ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );	//start
    
    
    ////----------HOST MEMORY ALLOC----------
    
    /* const variable */
    HANDLE_ERROR( hipHostAlloc( (void**)&e_host, sizeof(node_t)*2*E, hipHostMallocDefault ) );        //Edge
    HANDLE_ERROR( hipHostAlloc( (void**)&C_host, sizeof(flow_t)*N*N, hipHostMallocDefault ) );        //Capacity
    HANDLE_ERROR( hipHostAlloc( (void**)&D_host, sizeof(flow_t)*N*N, hipHostMallocDefault ) );        //Demand
    
    /* given variable */
    HANDLE_ERROR( hipHostAlloc( (void**)&w_host, sizeof(dist_t)*N*N, hipHostMallocDefault ) );         //Weight
    
    /* temporary variable */
    dist_t* d_host; HANDLE_ERROR( hipHostAlloc( (void**)&d_host, sizeof(dist_t)*N*N, hipHostMallocDefault ) );      //Distance
    flag_t* p_host; HANDLE_ERROR( hipHostAlloc( (void**)&p_host, sizeof(flag_t)*N*N, hipHostMallocDefault ) );      //Predtable
    flag_t* n_host; HANDLE_ERROR( hipHostAlloc( (void**)&n_host, sizeof(flag_t)*N*N, hipHostMallocDefault ) );      //Nexttable
    flow_t* f_host; HANDLE_ERROR( hipHostAlloc( (void**)&f_host, sizeof(flow_t)*E, hipHostMallocDefault ) );        //Flow
    
    //init on host
    for (node_t k = 0; k < E; k++) { e_host[k*2] = (node_t)edge[k][0];   e_host[k*2+1] = (node_t)edge[k][1];  }
    for (node_t i = 0; i < N; i++) { for (node_t j = 0; j < N; j++) { C_host[i*N+j] = (flow_t)capacity[i][j]; } }
    for (node_t i = 0; i < N; i++) { for (node_t j = 0; j < N; j++) { D_host[i*N+j] = (flow_t)distance[i][j]; } }
    for (node_t i = 0; i < N; i++) { for (node_t j = 0; j < N; j++) { w_host[i*N+j] = weightEdge[i*N+j]; } }
    for (node_t i = 0; i < N; i++) { for (node_t j = 0; j < N; j++) { d_host[i*N+j] = i==j ? 0 : 10000000; } }
    memset(p_host, 0, sizeof(flag_t)*N*N);
    memset(n_host, 0, sizeof(flag_t)*N*N);
    memset(f_host, 0, sizeof(flow_t)*E);
    
    //print
    //for (int i = 0; i < N; i++) { for (int j = 0; j < N; j++) { printf(" %d", d[i][j]); } printf("\n"); }
    //for (int i = 0; i < N; i++) { for (int j = 0; j < N; j++) { printf(" %u", p[i][j]); } printf("\n"); }
    //for (int i = 0; i < N; i++) { for (int j = 0; j < N; j++) { printf(" %d", ne[i][j]); } printf("\n"); }
    
    
    
    
    ////----------DEVICE MEMORY ALLOC and TRANSFER----------
#ifndef CONST
    node_t* e; HANDLE_ERROR( hipMalloc( (void**)&e, sizeof(node_t)*2*E ) );        //Edge      (1x2xE)     120
    flow_t* C; HANDLE_ERROR( hipMalloc( (void**)&C, sizeof(flow_t)*N*N ) );        //Capacity  (4xNxN)     900
    flow_t* D; HANDLE_ERROR( hipMalloc( (void**)&D, sizeof(flow_t)*N*N ) );        //Demand    (4xNxN)     900
    dist_t* w; HANDLE_ERROR( hipMalloc( (void**)&w, sizeof(dist_t)*N*N ) );        //Weight    (4xNxN)     900
#endif
    dist_t* d; HANDLE_ERROR( hipMalloc( (void**)&d, sizeof(dist_t)*N*N ) );        //Distance  (4xNxN)     900
    flag_t* p; HANDLE_ERROR( hipMalloc( (void**)&p, sizeof(flag_t)*N*N ) );        //Predtable (8xNxN)     1800
    flag_t* n; HANDLE_ERROR( hipMalloc( (void**)&n, sizeof(flag_t)*N*N ) );        //Nexttable (8xNxN)     1800
    flow_t* f; HANDLE_ERROR( hipMalloc( (void**)&f, sizeof(flow_t)*E ) );          //Flow      (4xE)       240
    
    
    
#ifndef CONST
    HANDLE_ERROR( hipMemcpy( e, e_host, sizeof(node_t)*2*E, hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( C, C_host, sizeof(flow_t)*N*N, hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( D, D_host, sizeof(flow_t)*N*N, hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( w, w_host, sizeof(dist_t)*N*N, hipMemcpyHostToDevice ) );
#else
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( e), e_host, sizeof(node_t)*2*E ) );
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( C), C_host, sizeof(flow_t)*N*N ) );
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( D), D_host, sizeof(flow_t)*N*N ) );
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( w), w_host, sizeof(dist_t)*N*N ) );
#endif
    HANDLE_ERROR( hipMemcpy( d, d_host, sizeof(dist_t)*N*N, hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemset( p, 0, sizeof(flag_t)*N*N ) );
    HANDLE_ERROR( hipMemset( n, 0, sizeof(flag_t)*N*N ) );
    HANDLE_ERROR( hipMemset( f, 0, sizeof(flow_t)*E ) );
    
    
    ////----------GET EDGE_PRED----------

    dim3 blocks(mp,1);
    dim3 threads(N/blocks.x+1, N);
    printf("block/thread = x(%d,%d) y(%d,%d)\n", blocks.x, threads.x, blocks.y, threads.y );
    
    for (edge_t k = 0; k < E; k++) {       //for each edge_t
        node_t x = e_host[k*2],  y = e_host[k*2+1];
#       ifdef __HIPCC__
        gpu_ecmpWarshallfloydEdge<<<blocks, threads>>>(k, x, y, d, p);
#       else
        gpuemulate(blocks,threads) gpu_ecmpWarshallfloydEdge(k, x, y, d_host, p_host);
#       endif
        
    }

    
    
#   if 1
    HANDLE_ERROR( hipEventRecord( wf, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( wf ) );
    HANDLE_ERROR( hipEventElapsedTime( &ms, start, wf ) );
    printf( "time: %f ms\n", ms );
#   ifdef __HIPCC__
    HANDLE_ERROR( hipMemcpy( p_host, p, sizeof(flag_t)*N*N, hipMemcpyDeviceToHost ) );
#   endif
    //printf("Pred Table : \n");      printTable(p_host);
#   endif
    

    
    return 0;
    
    
    
    
    
    
    
    
    
    
    
    
    ////----------GET EDGE_NEXT----------
    for (node_t i = 0; i < N; i++) {
        for (node_t j = 0; j < N; j++) {
            if (i == j) { continue; }
            n_host[i*N+j] = getNeighborEdges(p_host[i*N+j], i, p_host, e);
        }
    }
    
#   if 0
    printf("Next Table : \n");      printTable(n_host);
    HANDLE_ERROR( hipEventRecord( wf, 0 ) );
    HANDLE_ERROR( hipEventElapsedTime( &ms, start, wf ) );
    printf( "time: %f ms\n", ms );
#   endif
    
    
    ////----------TRACE AND ALLOC----------
    for(edge_t k = 0; k < E; k++ ){ f[k] = 0; }
    
    for (node_t i = 0; i < N; i++) {
        for (node_t j = 0; j < N; j++) {
            
            if (i == j) { continue; }
            traceAndAlloc(i, j, n_host, 1, D_host[i*N+j], f_host, e_host);
            
        }
    }
    
    
    ////----------GET L----------
    
    double l = 0.0;
    
    for(int k = 0; k < E; k++ ){
        
        int x = e_host[k*2],  y = e_host[k*2+1];
        double tmp = f_host[k] / capacity[x][y];
        
        if(l < tmp){ l = tmp; }
    }
    
    
    printf("L = %10f\n", l);
    
    
    
#   if 0
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );	//stop
    HANDLE_ERROR( hipEventSynchronize( stop ) );	//stop
    HANDLE_ERROR( hipEventElapsedTime( &ms, start, stop ) );
    printf( "time: %f ms\n", ms );
#   endif
    
    
    // device mem free
#ifndef CONST
    HANDLE_ERROR( hipFree( e ) );
    HANDLE_ERROR( hipFree( C ) );
    HANDLE_ERROR( hipFree( D ) );
    HANDLE_ERROR( hipFree( w ) );
#endif
    HANDLE_ERROR( hipFree( d ) );
    HANDLE_ERROR( hipFree( p ) );
    HANDLE_ERROR( hipFree( n ) );
    HANDLE_ERROR( hipFree( f ) );
    
    // host mem free
    HANDLE_ERROR( hipHostFree( e_host ) );
    HANDLE_ERROR( hipHostFree( C_host ) );
    HANDLE_ERROR( hipHostFree( D_host ) );
    HANDLE_ERROR( hipHostFree( w_host ) );
    HANDLE_ERROR( hipHostFree( d_host ) );
    HANDLE_ERROR( hipHostFree( p_host ) );
    HANDLE_ERROR( hipHostFree( n_host ) );
    HANDLE_ERROR( hipHostFree( f_host ) );
    
    
    
    //time
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( wf ) );
    
    
    
    
    
}











